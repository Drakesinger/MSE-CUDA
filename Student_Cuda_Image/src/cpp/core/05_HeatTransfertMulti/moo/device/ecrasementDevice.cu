#include "hip/hip_runtime.h"
#include <iostream>

#include "Indice2D.h"
#include "IndiceTools.h"
#include "cudaTools.h"
#include "Device.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void ecrasement(float* ptrImageInOutput, float* ptrImageHeater, float* ptrImageOutput, int w, int h);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__global__ void ecrasement(float* ptrImageInOutput, float* ptrImageHeater, float* ptrImageOutput, int w, int h)
    {
    const int WH = w * h;

    const int NB_THREAD = Indice2D::nbThread();
    const int TID = Indice2D::tid();

    int s = TID;

    while (s < WH)
	{

	bool t = ptrImageHeater[s] != 0;

	ptrImageOutput[s] = t * ptrImageHeater[s] + (1-t) * ptrImageInOutput[s];

	s += NB_THREAD;
	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

