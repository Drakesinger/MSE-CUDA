#include "hip/hip_runtime.h"
#include "MandelbrotJuliaMultiGPU.h"

#include <assert.h>

#include "Device.h"
#include "MathTools.h"

using cpu::IntervalI;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void mandelbrot(uchar4* ptrDevPixels, int w, int h, DomaineMath domaineMath, int n);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

MandelbrotJuliaMultiGPU::MandelbrotJuliaMultiGPU(int w, int h, int nMin, int nMax) : variateurN(IntervalI(nMin, nMax), 1)
{
    // Inputs
    this->w = w;
    this->h = h;

    this->ptrDomaineMathInit = new DomaineMath(-2.1, -1.3, 0.8, 1.3); // Mandelbrot
    // this->ptrDomaineMathInit = new DomaineMath(-1.3, -1.4, 1.4, 1.3); // Julia

    // Tools
    this->dg = dim3(8, 8, 1); // disons a optimiser
    this->db = dim3(16, 16, 1); // disons a optimiser
    this->deviceID = Device::getDeviceId();
    this->deviceIDBottom = 5;
    this->size = sizeof(uchar4) * w * (h / 2);

    //Outputs
    this->title = "MandelbrotJulia Multi GPU";

    HANDLE_ERROR(hipSetDevice(deviceIDBottom));
    HANDLE_ERROR(hipMalloc(&ptrDevTab1, size));
    HANDLE_ERROR(hipMemset(ptrDevTab1, 0, size));
    HANDLE_ERROR(hipSetDevice(deviceID));

    // Check:
    Device::assertDim(dg, db);
}

MandelbrotJuliaMultiGPU::~MandelbrotJuliaMultiGPU()
{
    HANDLE_ERROR(hipSetDevice(deviceIDBottom));
    HANDLE_ERROR(hipFree(ptrDevTab1));
    HANDLE_ERROR(hipSetDevice(deviceID));

    delete ptrDomaineMathInit;
}

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 */
void MandelbrotJuliaMultiGPU::process(uchar4* ptrDevPixels0, int w, int h, const DomaineMath& domaineMath)
{
    DomaineMath dmTop = domaineMath;
    dmTop.y1 = domaineMath.y1 / 2;

    DomaineMath dmBottom = domaineMath;
    dmBottom.y0 = domaineMath.y0 + (domaineMath.y1 - domaineMath.y0) / 2;

    #pragma omp parallel sections
    {
	#pragma omp section
	{
	    mandelbrot<<<dg,db>>>(ptrDevPixels0,w,h/2,dmTop, n);
	}

	#pragma omp section
	{
	    int deviceID = Device::getDeviceId();
	    HANDLE_ERROR(hipSetDevice(deviceIDBottom));

	    this->ptrDevBottomImage0 = ptrDevPixels0 + (w * (h / 2));

	    // kernel
	    mandelbrot<<<dg,db>>>(ptrDevTab1,w,h/2,dmBottom, n);

	    // MM copie sur device0 (affichage)
	    HANDLE_ERROR(hipMemcpy(ptrDevBottomImage0, ptrDevTab1, size, hipMemcpyDeviceToDevice));
	    HANDLE_ERROR(hipSetDevice(deviceID));
	}
    }
}

/**
 * Override
 * Call periodicly by the API
 */
void MandelbrotJuliaMultiGPU::animationStep()
{
    this->n = variateurN.varierAndGet(); // in [0,2pi]
}

/*--------------*\
 |*	get	 *|
 \*--------------*/

/**
 * Override
 */
DomaineMath* MandelbrotJuliaMultiGPU::getDomaineMathInit(void)
{
    return ptrDomaineMathInit;
}

/**
 * Override
 */
float MandelbrotJuliaMultiGPU::getAnimationPara(void)
{
    return n;
}

/**
 * Override
 */
int MandelbrotJuliaMultiGPU::getW(void)
{
    return w;
}

/**
 * Override
 */
int MandelbrotJuliaMultiGPU::getH(void)
{
    return h;
}

/**
 * Override
 */
string MandelbrotJuliaMultiGPU::getTitle(void)
{
    return title;
}

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
