#include "hip/hip_runtime.h"

#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"

#include "RayTracingMath.h"
#include "IndiceTools.h"
#include "Sphere.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void raytracing(uchar4* ptrDevPixels, Sphere* ptrDevSphere, int nbSphere, int w, int h, float t);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__global__ void raytracing(uchar4* ptrDevPixels, Sphere* ptrDevSphere, int nbSphere, int w, int h, float t)
    {
    RayTracingMath rayTracingMath = RayTracingMath(t); // ici pour preparer cuda

    const int WH = w * h;

    const int NB_THREAD = Indice2D::nbThread();
    const int TID = Indice2D::tid();

    int s = TID;

    int i;
    int j;
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &i, &j);
	rayTracingMath.colorXY(&ptrDevPixels[s],ptrDevSphere,nbSphere, i,j,t);
	s += NB_THREAD;
	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

